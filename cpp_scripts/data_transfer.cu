#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void access_data(int* input){
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    printf("tid: %d, gid: %d, value: %d \n", threadIdx.x, gid, input[gid]);
}


int main(){

    int size = 128;
    int byte_size = size * sizeof(int);

    int* h_input = (int*)malloc(byte_size);

    // initialize the input
    for(int i=0; i<size; ++i){
        h_input[i] = i;
    }

    // need memory allocated in the device
    // also
    int* d_input;

    hipMalloc((void**)& d_input, byte_size);

    // now transfer the data to the device
    hipMemcpy(d_input, h_input, 
    byte_size, hipMemcpyHostToDevice);

    dim3 block(64);
    dim3 grid(2);

    access_data<<<grid, block>>>(d_input);
    hipDeviceSynchronize();


    // reclaim the memory
    hipFree(d_input);
    free(h_input);

    hipDeviceReset();
    return 0;

}