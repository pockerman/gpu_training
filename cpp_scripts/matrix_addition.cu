#include <iostream>
#include <hip/hip_runtime.h>

#define N 64


// Kernel definition
__global__ void mat_addition(float* A[N], float* B[N],
                             float* C[N])
{
    int i = threadIdx.x;
    int j = threadIdx.y;
    C[i][j] = A[i][j] + B[i][j];
}


int main(){

const int N_ROWS = 64;
const int N_COLS = 64;

float hA[N][N];
float hB[N][N];
float hC[N][N];

for(int r=0; r<N; ++r){
  for(int c=0; c<N; ++c){
  
  	if (r == c){
  		hA[r][c] = r+c;
  		hB[r][c] = r+c;
  	}
  	else{
  		hA[r][c] = 0.0;
  		hB[r][c] = 0.0;
  	}
  	
  	hC[r][c] = 0.0;	 
  }
}


float* dA[N];
float* dB[N];
float* dC[N]; 

// allocate memory on the device

hipMalloc((void**)&dA, (N*N)*sizeof(float));
hipMalloc((void**)&dB, (N*N)*sizeof(float));
hipMalloc((void**)&dC, (N*N)*sizeof(float));

hipMemcpy(dA, hA, (N*N)*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(dB, hB, (N*N)*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(dC, hC, (N*N)*sizeof(float), hipMemcpyHostToDevice);

int numBlocks = 1;
dim3 threadsPerBlock(N, N);
mat_addition<<<numBlocks,threadsPerBlock>>>(dA, dB, dC);

hipMemcpy(hC, dC, (N*N)*sizeof(float), hipMemcpyDeviceToHost);

hipFree(dA); 
hipFree(dB); 
hipFree(dC);

return 0;
}


