
#include <hip/hip_runtime.h">
#include <vector>

namespace cutils{

__device__ std::vector<int> get_thread_global_coords(){

	std::vector<int> ids(3, -1);
	ids[0] = blockIdx.x * blockDim.x + threadIdx.x ;
	ids[1] = blockIdx.y * blockDim.y + threadIdx.y ;
	ids[2] = blockIdx.z * blockDim.z + threadIdx.z ;
	return ids;

}

}
