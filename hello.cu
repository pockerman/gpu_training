#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

#include "cuda_utils.h"


__global__ void hello(){

	auto t_ids = cutils::get_thread_global_coords();
	std::cout<<"x "<<t_ids[0]<<std::endl;
	std::cout<<"y "<<t_ids[1]<<std::endl;
	std::cout<<"z "<<t_ids[2]<<std::endl;
	
}

int main(){

hello<<1, 10>>();
hipDeviceSynchronize();

return 0; 
}
