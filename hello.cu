#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello(){

	printf("Hello world\n");
}

int main(){

hello<<<1, 10>>>();

// wait until all threads execute
hipDeviceSynchronize();

//reset the device
hipDeviceReset();

return 0; 
}
