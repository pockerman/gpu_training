#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

#define N 64


// Kernel definition
__global__ void mat_addition(float A[N][N], float B[N][N],
                             float C[N][N])
{
    int i = threadIdx.x;
    int j = threadIdx.y;
    C[i][j] = A[i][j] + B[i][j];
}


int main(){

const int N_ROWS = 64;
const int N_COLS = 64;

float hA[N][N];
float hB[N][N];
float hC[N][N];

for(int r=0; r<N; ++r){
  for(int c=0; c<N; ++c){
  
  	if (r == c){
  		hA[r][c] = r+c;
  		hB[r][c] = r+c;
  	}
  	else{
  		hA[r][c] = 0.0;
  		hB[r][c] = 0.0;
  	}
  	
  	hC[r][c] = 0.0;	 
  }
}


float* dA = NULL;
float* dB = NULL;
float* dC = NULL; 

// allocate memory on the device

hipMalloc((void**)&dA, (N*N_COLS)*sizeof(float));
hipMalloc((void**)&dB, (N*N_COLS)*sizeof(float));
hipMalloc((void**)&dC, (N*N_COLS)*sizeof(float));

hipMemcpy(dA, A, (N_ROWS*N_COLS)*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(dB, B, (N_ROWS*N_COLS)*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(dC, C, (N_ROWS*N_COLS)*sizeof(float), hipMemcpyHostToDevice);

int numBlocks = 1;
dim3 threadsPerBlock(N_ROWS, N_ROWS);
mat_addition<<<numBlocks,threadsPerBlock>>>(dA, dB, dC);

hipMemcpy(hC, dC, (N_ROWS*N_COLS)*sizeof(float), hipMemcpyDeviceToHost);

hipFree(dA); 
hipFree(dB); 
hipFree(dC);

return 0;
}


