#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

__global__ void hello(){

	printf("Hello world\n");
}


void query_device()
{
    int device_count = 0;
    hipGetDeviceCount(&device_count);

    if(device_count == 0){
        std::cout<<"No CUDA capable device found"<<std::endl;
        return;
    }

    int device_no = 0;
    hipDeviceProp_t iprop;
    hipGetDeviceProperties(&iprop, device_no);

    std::cout<<"Device number: "<<device_no<<" property name: "<<iprop.name<<std::endl;
    std::cout<<"Number of multiprocessors: "<<iprop.multiProcessorCount<<std::endl;
    std::cout<<"Clock rate: "<<iprop.clockRate<<std::endl;
}

int main(){

query_device();

// wait until all threads execute
hipDeviceSynchronize();

//reset the device
hipDeviceReset();

return 0; 
}