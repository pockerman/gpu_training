#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void square(float* d_out, float* d_in){

// the thread id
int idx = threadIdx.x;
float f = d_in[idx];
d_out[idx] = f * f;

}

int main(int argc, char** argv){
const int ARRAY_SIZE = 64;
const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

// generate input array on the host
float h_int[ARRAY_SIZE];
for(int i=0; i<ARRAY_SIZE; ++i){
   h_in[i] = float(i);
}

float h_out[ARRAY_SIZE];

float* d_in = NULL;
float* d_out = NULL;

// allocate GPU memory
hipMalloc((void **) &d_in, ARRAY_BYTES);
hipMalloc((void **) &d_out, ARRAY_BYTES);

// transfer array to GPU 
hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

// launch kernel
square<<<1, ARRAY_SIZE>>>(d_out, d_in);

hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);


for(int i=0; i<ARRAY_SIZE; ++i){
	std::cout<<h_in[i]<<", "<<h_out[i]<<std::endl;
}

// free memory
hipFree(d_in);
hipFree(d_out);


return 0;
}
